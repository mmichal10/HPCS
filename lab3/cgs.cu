#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate graident solver on GPU
 * using CUBLAS and CUSPARSE
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas and cusparse */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA SDK samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and intialization

const char *sSDKname     = "conjugateGradient";

double mclock(){
     struct timeval tp;

     double sec,usec;
     gettimeofday( &tp, NULL );
     sec    = double( tp.tv_sec );
     usec   = double( tp.tv_usec )/1E6;
     return sec + usec;
}


#define dot_BS     32
#define kernel_BS  32

__global__
void saxpy(int n, float alpha, float *x, float *y) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i < n)
		y[i] = alpha * x[i] + y[i];
}

__global__
void scal(int n, float alpha, float *y) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i < n)
		y[i] = alpha * y[i];
}

__global__
void cpy(int n, float *src, float *dst) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i < n)
		dst[i] = src[i];
}
/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz)
{
    double RAND_MAXi = 1e6;
    double val_r     = 12.345 * 1e5;
    
    I[0] = 0, J[0] = 0, J[1] = 1;
    val[0] = (float)val_r/RAND_MAXi + 10.0f;
    val[1] = (float)val_r/RAND_MAXi;
    int start;

    for (int i = 1; i < N; i++)
    {
        if (i > 1)
        {
            I[i] = I[i-1]+3;
        }
        else
        {
            I[1] = 2;
        }

        start = (i-1)*3 + 2;
        J[start] = i - 1;
        J[start+1] = i;

        if (i < N-1)
        {
            J[start+2] = i + 1;
        }

        val[start] = val[start-1];
        val[start+1] = (float)val_r/RAND_MAXi + 10.0f;

        if (i < N-1)
        {
            val[start+2] = (float)val_r/RAND_MAXi;
        }
    }

    I[N] = nz;
}


void cgs_basic(int argc, char **argv, int N, int M){

    //int M = 0, N = 0, 
    int nz = 0, *I = NULL, *J = NULL;
    float *val = NULL;
    const float tol = 1e-10f;
    const int max_iter = 1000;
    float *x;
    float *rhs;
    float a, b, na, r0, r1;
    int *d_col, *d_row;
    float *d_val, *d_x, dot;
    float *d_r, *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp;
    int devID = findCudaDevice(argc, (const char **)argv);

    if (devID < 0)
    {
        printf("exiting...\n");
        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    int version = (deviceProp.major * 0x10 + deviceProp.minor);

    if (version < 0x11)
    {
        printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKname);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    /* Generate a random tridiagonal symmetric matrix in CSR format */
    //M = N = 32*64;//10; //1048576;
    printf("M = %d, N = %d\n", M, N);
    nz = (N-2)*3 + 4;
    I = (int *)malloc(sizeof(int)*(N+1));
    J = (int *)malloc(sizeof(int)*nz);
    val = (float *)malloc(sizeof(float)*nz);
    genTridiag(I, J, val, N, nz);

    /*
    for (int i = 0; i < nz; i++){
        printf("%d\t", J[i]);
    }
    printf("\n");
    for (int i = 0; i < nz; i++){
        printf("%2f\t", val[i]);
    }
    */

    x = (float *)malloc(sizeof(float)*N);
    rhs = (float *)malloc(sizeof(float)*N);

    for (int i = 0; i < N; i++)
    {
        rhs[i] = 1.0;
        x[i] = 0.0;
    }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    checkCudaErrors(cusparseStatus);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);

    checkCudaErrors(cusparseStatus);

    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;


    double t_start = mclock();
    hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);

    //hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);                                // PODMIEN FUNCKJE (I)
	saxpy<<<(N+255)/256, 256>>>(N, alpham1, d_Ax, d_r);
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                        // PODMIEN FUNCKJE (II)

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            //hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);                        // PODMIEN FUNCKJE (I)
			scal<<<(N+255)/256, 256>>>(N, b, d_p);
            //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);            // PODMIEN FUNCKJE (I)
			saxpy<<<(N+255)/256, 256>>>(N, alpha, d_r, d_p);
        }
        else
        {
            hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);                    // PODMIEN FUNCKJE (I)
        }

        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax); // PODMIEN FUNCKJE (III)
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);                  // PODMIEN FUNCKJE (II)
        a = r1 / dot;

        //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);                    // PODMIEN FUNCKJE (I)
		saxpy<<<(N+255)/256, 256>>>(N, a, d_p, d_x);
        na = -a;
        //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);                  // PODMIEN FUNCKJE (I)
		saxpy<<<(N+255)/256, 256>>>(N, na, d_Ax, d_r);

        r0 = r1;
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                    // PODMIEN FUNCKJE (II)
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }
    printf("TIME OF CGS_BASIC = %f\n", mclock() - t_start);

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

    float rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++)
    {
        rsum = 0.0;

        for (int j = I[i]; j < I[i+1]; j++)
        {
            rsum += val[j]*x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err)
        {
            err = diff;
        }
    }

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    free(I);
    free(J);
    free(val);
    free(x);
    free(rhs);
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

    hipDeviceReset();

    printf("Test Summary:  Error amount = %e\n", err);
    //exit((k <= max_iter) ? 0 : 1);


}
void cgs_TODO(int argc, char **argv, int N, int M){

    //int M = 0, N = 0, 
    int nz = 0, *I = NULL, *J = NULL;
    float *val = NULL;
    const float tol = 1e-10f;
    const int max_iter = 1000;
    float *x;
    float *rhs;
    float a, b, na, r0, r1;
    int *d_col, *d_row;
    float *d_val, *d_x, dot;
    float *d_r, *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp;
    int devID = findCudaDevice(argc, (const char **)argv);

    if (devID < 0)
    {
        printf("exiting...\n");
        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    int version = (deviceProp.major * 0x10 + deviceProp.minor);

    if (version < 0x11)
    {
        printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKname);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    /* Generate a random tridiagonal symmetric matrix in CSR format */
    //M = N = 32*64;//10; //1048576;
    printf("M = %d, N = %d\n", M, N);
    nz = (N-2)*3 + 4;
    I = (int *)malloc(sizeof(int)*(N+1));
    J = (int *)malloc(sizeof(int)*nz);
    val = (float *)malloc(sizeof(float)*nz);
    genTridiag(I, J, val, N, nz);

    /*
    for (int i = 0; i < nz; i++){
        printf("%d\t", J[i]);
    }
    printf("\n");
    for (int i = 0; i < nz; i++){
        printf("%2f\t", val[i]);
    }
    */

    x = (float *)malloc(sizeof(float)*N);
    rhs = (float *)malloc(sizeof(float)*N);

    for (int i = 0; i < N; i++)
    {
        rhs[i] = 1.0;
        x[i] = 0.0;
    }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    checkCudaErrors(cusparseStatus);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);

    checkCudaErrors(cusparseStatus);

    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;


    // sparse matrix vector product: d_Ax = A * d_x
    hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);  // PODMIEN FUNCKJE (ZADANIE-I)


    //azpy: d_r = d_r + alpham1 * d_Ax
    //hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);        			    // PODMIEN FUNCKJE (ZADANIE-I)
	saxpy<<<(N+255)/256, 256>>>(N, alpham1, d_Ax, d_r);
    //dot:  r1 = d_r * d_r
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                        // PODMIEN FUNCKJE (ZADANIE-III)

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
	    //scal: d_p = b * d_p
            //hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);                        // PODMIEN FUNCKJE (ZADANIE-I)
			scal<<<(N+255)/256, 256>>>(N, b, d_p);
	    //axpy:  d_p = d_p + alpha * d_r
            //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);            // PODMIEN FUNCKJE (ZADANIE-I)
			saxpy<<<(N+255)/256, 256>>>(N, alpha, d_r, d_p);
        }
        else
        {
            //cpy: d_p = d_r
            //hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);                    // PODMIEN FUNCKJE (ZADANIE-I)
			cpy<<<(N+255)/256, 256>>>(N, d_r, d_p);
        }

        //sparse matrix-vector product: d_Ax = A * d_p
        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax); // PODMIEN FUNCKJE (ZADANIE-II)
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);                  // PODMIEN FUNCKJE (ZADANIE-III)
        a = r1 / dot;

        //axpy: d_x = d_x + a*d_p
        //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);                    // PODMIEN FUNCKJE (ZADANIE-I)
		saxpy<<<(N+255)/256, 256>>>(N, a, d_p, d_x);
        na = -a;
	 
        //axpy:  d_r = d_r + na * d_Ax
        //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);                  // PODMIEN FUNCKJE (ZADANIE-I)
		saxpy<<<(N+255)/256, 256>>>(N, na, d_Ax, d_r);

        r0 = r1;
	
        //dot: r1 = d_r * d_r
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                    // PODMIEN FUNCKJE (ZADANIE-III)
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

    float rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++)
    {
        rsum = 0.0;

        for (int j = I[i]; j < I[i+1]; j++)
        {
            rsum += val[j]*x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err)
        {
            err = diff;
        }
    }

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    free(I);
    free(J);
    free(val);
    free(x);
    free(rhs);
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

    hipDeviceReset();

    printf("Test Summary:  Error amount = %e\n", err);
    //exit((k <= max_iter) ? 0 : 1);


}







int main(int argc, char **argv)
{
    //int N = 1e6;//1 << 20;
    //int N = 256 * (1<<10)  -10 ; //1e6;//1 << 20;
    int N = 1e5;
    int M = N; 
    
    cgs_basic(argc, argv, N, M);
    
    cgs_TODO(argc, argv, N, M);
}
